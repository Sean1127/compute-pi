#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
extern "C" {
#include "montecarlopi.h"
}

#ifndef ITERATIONSPERTHREAD
#define ITERATIONSPERTHREAD 4000
#endif

extern "C"
__global__ void monte_carlo_kernel( hiprandState* state, unsigned int seed, int *numbers)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    float x, y;

    hiprand_init(seed, index, 0, &state[index]);

    for(int i = 0; i < ITERATIONSPERTHREAD; i++) {
        x = hiprand_uniform (&state[index]);
        y = hiprand_uniform (&state[index]);
        sum += (x*x + y*y <= 1.0f);
    }
    numbers[index] = sum;
}

double compute_pi_montecarlo_gpu(size_t N)
{
    int threadsPerBlock = 1000;
    int blocksPerGrid = (N / threadsPerBlock) / ITERATIONSPERTHREAD;
    hiprandState *devStates;
    int *dev_nums;
    int *host_nums = (int *)malloc(sizeof(int) * threadsPerBlock * blocksPerGrid);
    if(host_nums == NULL)
        return 0;

    // malloc memory in gpu
    hipMalloc((void **)&devStates, sizeof(hiprandState) * threadsPerBlock * blocksPerGrid);
    hipMalloc((void **)&dev_nums, sizeof(int) * threadsPerBlock * blocksPerGrid);
    monte_carlo_kernel <<< blocksPerGrid, threadsPerBlock>>> (devStates, time(NULL), dev_nums);
    // copy data from device to host
    hipMemcpy(host_nums, dev_nums, sizeof(int) * threadsPerBlock * blocksPerGrid, hipMemcpyDeviceToHost);

    int total_in_quadcircle = 0;
    for(int i = 0; i < threadsPerBlock * blocksPerGrid; i++){
        total_in_quadcircle += host_nums[i];
    }
    double pi = 4 * ((double)total_in_quadcircle / N);

    hipFree(devStates);
    hipFree(dev_nums);

    return pi;
}

double compute_pi_montecarlo_cpu(size_t N)
{
    double pi = 0.0;
    size_t sum = 0;
    srand(time(NULL));
    for(size_t i = 0; i < N; i++)
    {
        double x = (double) rand() / RAND_MAX;
        double y = (double) rand() / RAND_MAX;
        if((x * x + y * y) < 1) {
            sum++; 
        }
    }
    pi = 4 * ((double)sum / N);
    return pi;
}
